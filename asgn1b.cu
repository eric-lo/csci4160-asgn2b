#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

#include"util.h"

// If you have referenced to any source code that is not written by you
// You have to cite them here.




extern "C" int asgn1b(Point * points, Point ** pPermissiblePoints, int number, int dim, int gpuid)
{
    // points -- input data
    // pPermissiblePoints -- your computed answer
    // number -- number of points in dataset
    // dim -- the dimension of the dataset
    // gpuid -- the gpu used to run the program
    
	int permissiblePointNum = 0;
	Point * permissiblePoints = NULL;

	hipSetDevice(gpuid);

	/**********************************************************************************
	 * Work here
	 * *******************************************************************************/

	
	*pPermissiblePoints = permissiblePoints;
	return permissiblePointNum;
}
